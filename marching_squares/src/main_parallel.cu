#include "hip/hip_runtime.h"

#include "gdal.h"
#include "cpl_conv.h" // for CPLMalloc()
#include <errno.h>

#include <stdio.h>
#include <stdlib.h>
#include <string>

#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/cudev.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace cv;

const char* FILENAME = "../data/cdnh44o.tif";  // File Location
const int CSZ = 15;                            // Cell Size
const bool DISPLAY = 0;                        // Display Contours
const bool ANIMATE = 0;                        // Animate Contours
const bool SAVE = 0;                           // Save Contour Image
const float ISOVALUE = 0;                      // Minimum Elevation of contour visible (0 - everything is visible, 50 - ) TO-DO
const float numContours = 1000;                // Num of different contour levels to show in output image

float adfMinMax[2];
int N = 2000, M = 2000;                                                                              
Mat img = Mat::zeros((N - 1) * CSZ, (M - 1) * CSZ, CV_8UC1);

void readDem(float* data)
{
    GDALDatasetH hDataset;
    GDALAllRegister();
    const GDALAccess eAccess = GA_ReadOnly;
    hDataset = GDALOpen(FILENAME, eAccess);
    if (hDataset == NULL)
    {
        printf("Error in loading Raster File\n");
        return;
    }

    GDALRasterBandH hBand;
    hBand = GDALGetRasterBand(hDataset, 1);
    int bGotMin, bGotMax;
    adfMinMax[0] = GDALGetRasterMinimum(hBand, &bGotMin);
    adfMinMax[1] = GDALGetRasterMaximum(hBand, &bGotMax);
    if (!(bGotMin && bGotMax))
        GDALComputeRasterMinMax(hBand, TRUE, (double*)adfMinMax);

    float *pafScanline;
    int nXSize = GDALGetRasterBandXSize(hBand);
    int nYSize = GDALGetRasterBandYSize(hBand);

    if(N > nYSize || M > nXSize)
    {
        N = nYSize;
        M = nXSize;
    }
    pafScanline = (float *)CPLMalloc(sizeof(float) * nXSize);

    for (int i = 0; i < N; i++)
    {
        CPLErr error = GDALRasterIO(hBand, GF_Read, 0, i, nXSize, 1,
                    pafScanline, nXSize, 1, GDT_Float32,
                    0, 0);
        for (int j = 0; j < M; j++)
        {
            data[i * M + j] = pafScanline[j];
            if(i == 0 && j == 0)
            {
                adfMinMax[0] = data[i * M + j];
                adfMinMax[1] = data[i * M + j];
            }
            adfMinMax[0] = std::min(data[i * M + j], adfMinMax[0]);
            adfMinMax[1] = std::max(data[i * M + j], adfMinMax[1]);
        }
    }

    CPLFree(pafScanline);
    GDALClose(hDataset);

}

__global__ void drawContours(short* contourGrid, cv::cudev::PtrStepSz<uchar> img, int N, int M)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int chunkRow = N / blockDim.y, chunkCol = M / blockDim.x;
    for(int i = row; i < row + chunkRow && i < N - 1; i++)
    {
        for(int j = col; j < col + chunkCol && j < M - 1; j++)
        {
            if(contourGrid[i * (M - 1) + j] == 1 || contourGrid[i * (M - 1) + j] == 14)
            {
                line(img(Rect(j * CSZ, i * CSZ, CSZ, CSZ)), Point(0, CSZ/2), Point(CSZ/2, CSZ), Scalar(255), 2, LINE_8);
            }
            else if(contourGrid[i * (M - 1) + j] == 2 || contourGrid[i * (M - 1) + j] == 13)
            {
                line(img(Rect(j * CSZ, i * CSZ, CSZ, CSZ)), Point(CSZ/2, CSZ), Point(CSZ, CSZ/2), Scalar(255), 2, LINE_8);
            }
            else if(contourGrid[i * (M - 1) + j] == 3 || contourGrid[i * (M - 1) + j] == 12)
            {
                line(img(Rect(j * CSZ, i * CSZ, CSZ, CSZ)), Point(0, CSZ/2), Point(CSZ, CSZ/2), Scalar(255), 2, LINE_8);
            }
            else if(contourGrid[i * (M - 1) + j] == 4 || contourGrid[i * (M - 1) + j] == 11)
            {
                line(img(Rect(j * CSZ, i * CSZ, CSZ, CSZ)), Point(CSZ/2, 0), Point(CSZ, CSZ/2), Scalar(255), 2, LINE_8);
            }
            else if(contourGrid[i * (M - 1) + j] == 6 || contourGrid[i * (M - 1) + j] == 9)
            {
                line(img(Rect(j * CSZ, i * CSZ, CSZ, CSZ)), Point(CSZ/2, 0), Point(CSZ/2, CSZ), Scalar(255), 2, LINE_8);
            }
            else if(contourGrid[i * (M - 1) + j] == 7 || contourGrid[i * (M - 1) + j] == 8)
            {
                line(img(Rect(j * CSZ, i * CSZ, CSZ, CSZ)), Point(0, CSZ/2), Point(CSZ/2, 0), Scalar(255), 2, LINE_8);
            }
            else if(contourGrid[i * (M - 1) + j] == 5)
            {
                line(img(Rect(j * CSZ, i * CSZ, CSZ, CSZ)), Point(0, CSZ/2), Point(CSZ/2, 0), Scalar(255), 2, LINE_8);
                line(img(Rect(j * CSZ, i * CSZ, CSZ, CSZ)), Point(CSZ/2, CSZ), Point(CSZ, CSZ/2), Scalar(255), 2, LINE_8);
            }
            else if(contourGrid[i * (M - 1) + j] == 10)
            {
                line(img(Rect(j * CSZ, i * CSZ, CSZ, CSZ)), Point(CSZ/2, 0), Point(CSZ, CSZ/2), Scalar(255), 2, LINE_8);
                line(img(Rect(j * CSZ, i * CSZ, CSZ, CSZ)), Point(0, CSZ/2), Point(CSZ/2, CSZ), Scalar(255), 2, LINE_8);
            }
        }
    }
}

__global__ void makeContourGrid(short* contourGrid, const float* data, int N, int M, int isovalue = ISOVALUE)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int chunkRow = N / blockDim.y, chunkCol = M / blockDim.x;
    for(int i = row; i < row + chunkRow && i < N - 1; i++)
        for(int j = col; j < col + chunkCol && j < M - 1; j++)
                contourGrid[i * (M - 1) + j] = ((data[i * M + j] > isovalue) * 8) + ((data[i * M + j + 1] > isovalue) * 4) + ((data[(i + 1) * M + j + 1] > isovalue) * 2) + ((data[(i + 1) * M + j] > isovalue) * 1);
}

int main(int argc, char* argv[])
{

    // loading data in host memory

    float *h_A = (float*)malloc(sizeof(float) * N * M);
    readDem(h_A);
    float stepSz = (adfMinMax[1] - std::max(ISOVALUE, adfMinMax[0])) / numContours;
    printf("Max: %f, Min: %f\n", adfMinMax[1], adfMinMax[0]);
    printf("Step Size: %f\n", stepSz);
    short *contourGrid = (short*)malloc(sizeof(short) * (N - 1) * (M - 1));


    hipError_t err = hipSuccess;

    // device memory allocation
    float* d_A = NULL;
    err = hipMalloc((void**)&d_A, sizeof(float) * N * M);
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device DEM array (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    short* d_cg = NULL;
    err = hipMalloc((void**)&d_cg, sizeof(short) * (N - 1) * (M - 1));
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device Contour Grid (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // uint8_t *d_img = NULL;
    // err = hipMalloc((void**)&d_img, sizeof(uint8_t) * ((N - 1) * CSZ) * ((M - 1) * CSZ));
    // if(err != hipSuccess)
    // {
    //     fprintf(stderr, "Failed to allocate device Contour image (error code %s)!\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    cuda::GpuMat d_img;

    // host to device data transfer
    err = hipMemcpy(h_A, d_A, sizeof(float) * N * M, hipMemcpyHostToDevice);
    // err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to transfer data from h_A to d_A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(contourGrid, d_cg, sizeof(short) * (N - 1) * (M - 1), hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to transfer data from contourGrid to d_cg (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    d_img.upload(img);
    // err = hipMemcpy(img, d_img, sizeof(img), hipMemcpyHostToDevice);
    // if(err != hipSuccess)
    // {
    //     fprintf(stderr, "Failed to transfer data from img to d_img (error code %s)!\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    // kernel launch
    dim3 block(32, 32);
    dim3 grid(16, 16);

    for (float i = std::max(ISOVALUE, adfMinMax[0]); i < adfMinMax[1]; i+=stepSz)
    {
        makeContourGrid<<<block, grid>>>(d_cg, d_A, N, M, i);
        drawContours<<<block, grid>>>(d_cg, d_img, N, M);
        // if(ANIMATE)
        // {
        //     namedWindow("Contours", 0);
        //     imshow("Contours", img);
        //     waitKey(1);
        // }
    }

    img = d_img;
    free(h_A);
    free(contourGrid);

    if(DISPLAY)
    {
        namedWindow("Contours", 0);
        imshow("Contours", img);
        waitKey(0);
    }

    if(SAVE)
    {
      imwrite("Contour.png", img);
    }

    
    return 0;
}
